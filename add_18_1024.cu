#include <stdio.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void add(float *output, float *input1, float *input2)
{
  if (blockIdx.x < 18)
  {
    {
      if (threadIdx.x < 1024)
      {
        {
          int index = (blockIdx.x * 1024) + threadIdx.x;
          output[index] = input1[index] + input2[index];
        }
      }
    }
  }
}

extern "C" void add_kernel(float *C, float *A, float *B, int size) {
            float *d_A, *d_B, *d_C;

            hipMalloc(&d_A, size * sizeof(float));
            hipMalloc(&d_B, size * sizeof(float));
            hipMalloc(&d_C, size * sizeof(float));

            hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_B, B, size * sizeof(float), hipMemcpyHostToDevice);

            dim3 blockSize(1024);
            dim3 numBlocks(256);

            add<<<numBlocks, blockSize>>>(d_C, d_A, d_B);

            hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);
            }
        